#include "hip/hip_runtime.h"
!pip install git+https://github.com/andreinechaev/nvcc4jupyter.git
%load_ext nvcc_plugin
%%cu

%%cu
// Matrix multiply
// To run on colab:
//   !pip install git+https://github.com/andreinechaev/nvcc4jupyter.git
//   %load_ext nvcc_plugin
//   %%cu
// To compile from command line:
//   nvcc matrix_mult.cu -o matrix_mult

#include <stdio.h>
#include <stdlib.h>
#include <time.h>


// ==========================================================================
// ==========================================================================
// Kernel

__global__ void matrixMul(int* m1, int* m2, int* p, int n)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int column = blockIdx.x * blockDim.x + threadIdx.x;
  int p_sum = 0;
  for (int i = 0; i < n; i++) {
    p_sum += m1[row * n + i] * m2[i * n + column];
  }
  p[row * n + column] = p_sum;
}


void matrixMulSeq(int* m1, int* m2, int* p, int n)
{
  // int row = blockIdx.y * blockDim.y + threadIdx.y;
  // int column = blockIdx.x * blockDim.x + threadIdx.x;
  for (int row = 0; row < n; row++) {

    for (int column = 0; column < n; column++) {
      int p_sum = 0;
      for (int i = 0; i < n; i++) {
        p_sum += m1[row * n + i] * m2[i * n + column];
      }
      p[row * n + column] = p_sum;
    }

  }
}


// ==========================================================================
// ==========================================================================
// cuda_utils.cu


bool cmpMat2d(int* m1, int* m2, int r, int c)
{
  for (int i = 0; i < r; i++) {
    for (int j = 0; j < c; j++) {
      if (m1[i*c+j] != m2[i*c+j]) {return false;}
    }
  }
  return true;
}


double t1;
void startTimer()
{
  t1 = (double)clock()/CLOCKS_PER_SEC;
}


double stopTimer(char* title)
{
  double t2 = (double)clock()/CLOCKS_PER_SEC;
  double elapsedTime = t2 - t1;
  printf ("%s: Elapsed time = %6.6f\n", title, elapsedTime);
 return elapsedTime;
}


void printSlice(char* title, int* mat2d, int n, int r1, int c1, int height, int width)
{
  printf("%s\n", title);
  for (int i = r1; i < r1+height; i++) {
    for (int j = c1; j < c1+width; j++) {
      printf("[%d,%d]=%6d ", i, j, mat2d[i*n+j]);
    }
    printf("\n");
  }
 printf("\n");
}


// ==========================================================================
// ==========================================================================
// Main

int main()
{
  int n = 1 << 10;
  size_t bytes = n * n * sizeof(int);
  int* h_m1 = (int*)malloc(bytes);
  int* h_m2 = (int*)malloc(bytes);
  int* h_p = (int*)malloc(bytes);

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      h_m1[i*n + j] = rand() % 1024; // 2; // rand() % 1024;
      h_m2[i*n + j] = rand() % 1024; //3; // rand() % 1024;
    }
  }

  printSlice("M1", h_m1, n, 10, 20, 5, 5);
  printSlice("M2", h_m2, n, 10, 20, 5, 5);

  // ------------------------------------------------------------------------
  // Run on CPU
  int* h_p_cpu = (int*)malloc(bytes);
  startTimer();
  matrixMulSeq(h_m1, h_m2, h_p_cpu, n);
  double elapsedTimeCPU = stopTimer("CPU TIME");
  printSlice("Product (CPU))", h_p_cpu, n, 10, 20, 5, 5); 

  // ------------------------------------------------------------------------
  // Accelerate with GPU
  startTimer();
  int* d_m1;
  int* d_m2;
  int* d_p;
  hipMalloc(&d_m1, bytes);
  hipMalloc(&d_m2, bytes);
  hipMalloc(&d_p, bytes);
  hipMemcpy(d_m1, h_m1, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_m2, h_m2, bytes, hipMemcpyHostToDevice);
  int threads_per_block = 16;
  dim3 block_size (threads_per_block, threads_per_block);
  dim3 grid_size (n / block_size.x, n / block_size.y);
  matrixMul<<<grid_size, block_size >>>(d_m1, d_m2, d_p, n);
  hipMemcpy(h_p, d_p, bytes, hipMemcpyDeviceToHost);
  double elapsedTimeGPU = stopTimer("GPU TIME");
  printSlice("Product (GPU)", h_p, n, 10, 20, 5, 5);
  printf("GPU matmult done\n");

  // ------------------------------------------------------------------------
 // Wrap up
  double performanceIncrease =  elapsedTimeCPU / elapsedTimeGPU;
  printf("Performance Improvement = %3.2fx faster\n\n", performanceIncrease);


  if (cmpMat2d(h_p, h_p_cpu, n, n)) {
    printf("PASS: GPU == CPU\n");
  } else {
    printf("FAIL: GPU != CPU\n");
  }
  printf("CPU matmult done\n");

  return 0;
}

