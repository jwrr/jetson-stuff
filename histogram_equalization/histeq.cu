#include "hip/hip_runtime.h"
// This Program is based on Abubakr Shafique's (abubakr.shafique@gmail.com) program
// at https://github.com/abubakr-shafique/Histogram_Equalization_CUDA_CPP.
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;

// ============================================================================
// ============================================================================

__global__ void find_minmax_gpu(unsigned char* img, int N_CHAN, int* Min, int* Max)
{
  int x = blockIdx.x;
  int y = blockIdx.y;
  int pixel_id = (x + y * gridDim.x) * N_CHAN;
  for (int i = 0; i < N_CHAN; i++) {
    atomicMin(&Min[i], img[pixel_id + i]);
    atomicMax(&Max[i], img[pixel_id + i]);
  }
}


__device__ int new_pixel_value(int Value, int Min, int Max)
{
  int Target_Min = 0;
  int Target_Max = 255;
  return (Target_Min + (Value - Min) * (int)((Target_Max - Target_Min)/(Max - Min)));
}


__global__ void histeq_gpu(unsigned char* img, int N_CHAN, int* Min, int* Max)
{
  int x = blockIdx.x;
  int y = blockIdx.y;
  int pixel_id = (x + y * gridDim.x) * N_CHAN;
  for (int i = 0; i < N_CHAN; i++) {
    img[pixel_id + i] = new_pixel_value(img[pixel_id + i], Min[i], Max[i]);
  }
}

// ============================================================================
// ============================================================================

void histeq_wrapper(unsigned char* img, int Height, int Width, int N_CHAN)
{
  unsigned char* Dev_Image = NULL;
  int* Dev_Min = NULL;
  int* Dev_Max = NULL;
  
  //allocate cuda variable memory
  hipMalloc((void**)&Dev_Image, Height * Width * N_CHAN);
  hipMalloc((void**)&Dev_Min, N_CHAN * sizeof(int));
  hipMalloc((void**)&Dev_Max, N_CHAN * sizeof(int));
  
  int Min[3] = {255, 255, 255};
  int Max[3] = {0, 0, 0};
  
  //copy CPU data to GPU
  hipMemcpy(Dev_Image, img, Height * Width * N_CHAN, hipMemcpyHostToDevice);
  hipMemcpy(Dev_Min, Min, N_CHAN * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Dev_Max, Max, N_CHAN * sizeof(int), hipMemcpyHostToDevice);
  
  dim3 Grid_Image(Width, Height);
  find_minmax_gpu <<<Grid_Image, 1>>>(Dev_Image, N_CHAN, Dev_Min, Dev_Max);
  histeq_gpu <<<Grid_Image, 1>>>(Dev_Image, N_CHAN, Dev_Min, Dev_Max);
  
  //copy memory back to CPU from GPU
  hipMemcpy(img, Dev_Image, Height * Width * N_CHAN, hipMemcpyDeviceToHost);
  
  //free up the memory of GPU
  hipFree(Dev_Image);
}

// ============================================================================
// ============================================================================

int main()
{
  // image from Wikipedia: https://en.wikipedia.org/wiki/Histogram_equalization
  string img_path = "/home/jwrr/github/jetson-stuff/histogram_equalization/low_contrast.jpg";
  cv::Mat img_orig = cv::imread(img_path);
  cv::Mat img = cv::imread(img_path);
  if (img.empty()) {
    std::cout << "Error - Could not read image: " << img_path << std::endl;
    return 1;
  }
  cout << "Image Size: " << img.cols << "x" << img.rows << 
          ", Image Channels: " << img.channels() << endl;
  histeq_wrapper(img.data, img.rows, img.cols, img.channels());
  cv::imwrite("high_contrast.png", img);

  string windowName1 = "Low Contrast";
  cv::namedWindow(windowName1);
  cv::moveWindow(windowName1,100,100);
  cv::imshow(windowName1, img_orig);

  string windowName2 = "High Contrast";
  cv::namedWindow(windowName2);
  cv::moveWindow(windowName2,450,100);
  cv::imshow(windowName2, img);

  cv::waitKey(0); // Wait for any keystroke in the window
  cv::destroyWindow(windowName1);
  cv::destroyWindow(windowName2);
  return 0;
}

